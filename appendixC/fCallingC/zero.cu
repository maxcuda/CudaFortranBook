
#include <hip/hip_runtime.h>
/*
 *    Copyright (c) 2016, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.
 *
 *
 *   These example codes are a portion of the code samples from the companion
 *   website to the book "CUDA Fortran for Scientists and Engineers":
 *
 * http://store.elsevier.com/product.jsp?isbn=9780124169708
 *
 */

extern "C" __global__ void zero(float *a)
{
  a[blockIdx.x*blockDim.x+threadIdx.x] = 0.0;
}

